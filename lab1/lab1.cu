#include "hip/hip_runtime.h"
#include "lab1.h"
#include "PerlinNoise.h"
#include <cmath>
#include "stdio.h"
static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME = 240;


struct Lab1VideoGenerator::Impl {
	int t = 0;
};

Lab1VideoGenerator::Lab1VideoGenerator(): impl(new Impl) {
}

Lab1VideoGenerator::~Lab1VideoGenerator() {}

void Lab1VideoGenerator::get_info(Lab1VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};


void Lab1VideoGenerator::Generate(uint8_t *yuv) {

    
	// Create a PerlinNoise object with the reference permutation vector
	PerlinNoise pn;
	unsigned int kk = 0;
    unsigned int nn = 0;


    unsigned int rarray[240] = {0, 0, 0, 0, 0, 0, 85, 136, 187, 170, 204, 198, 170, 162, 68, 102, 193, 204, 230, 238, 221, 238, 153, 102, 0, 0, 0, 0, 0, 0, 0, 68, 85, 119, 165, 204, 255, 204, 153, 119, 85, 0, 0, 0, 0, 0, 0, 0, 119, 187, 255, 255, 255, 255, 255, 255, 136, 170, 255, 255, 255, 255, 255, 255, 204, 153, 51, 51, 51, 51, 51, 85, 85, 119, 153, 185, 233, 255, 255, 227, 210, 176, 159, 153, 153, 119, 102, 119, 119, 102, 187, 221, 255, 255, 255, 255, 255, 255, 221, 255, 255, 255, 255, 255, 255, 255, 238, 204, 153, 187, 170, 153, 204, 204, 204, 204, 209, 232, 240, 255, 255, 240, 232, 209, 204, 204, 204, 204, 153, 170, 187, 153, 204, 238, 255, 255, 255, 255, 255, 255, 255, 221, 255, 255, 255, 255, 255, 255, 221, 187, 102, 119, 119, 102, 119, 153, 153, 159, 176, 210, 227, 255, 255, 233, 185, 153, 119, 85, 85, 51, 51, 51, 51, 51, 153, 204, 255, 255, 255, 255, 255, 255, 170, 136, 255, 255, 255, 255, 255, 255, 187, 119, 0, 0, 0, 0, 0, 0, 0, 85, 119, 153, 204, 255, 204, 165, 119, 85, 68, 0, 0, 0, 0, 0, 0, 0, 102, 153, 238, 221, 238, 230, 204, 193, 102, 68, 162, 170, 198, 204, 170, 187, 136, 85, 0, 0, 0, 0, 0, 0 };
    unsigned int garray[240] = {60, 136, 170, 170, 170, 170, 170, 170, 187, 119, 102, 51, 0, 0, 68, 102, 0, 0, 63, 119, 170, 238, 221, 221, 221, 221, 221, 221, 159, 68, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 102, 187, 255, 255, 255, 255, 255, 255, 255, 187, 136, 85, 0, 0, 136, 170, 68, 51, 119, 170, 204, 255, 255, 255, 255, 255, 255, 255, 204, 153, 85, 68, 85, 79, 62, 62, 119, 142, 142, 136, 136, 153, 187, 221, 255, 255, 255, 255, 255, 255, 255, 221, 187, 164, 136, 136, 221, 255, 183, 204, 200, 221, 238, 255, 255, 255, 255, 255, 255, 255, 238, 221, 204, 187, 187, 204, 187, 179, 179, 187, 204, 187, 187, 204, 221, 238, 255, 255, 255, 255, 255, 255, 255, 238, 221, 200, 204, 183, 255, 221, 136, 136, 164, 187, 221, 255, 255, 255, 255, 255, 255, 255, 221, 187, 153, 136, 136, 142, 142, 119, 62, 62, 79, 85, 68, 85, 153, 204, 255, 255, 255, 255, 255, 255, 255, 204, 170, 119, 51, 68, 170, 136, 0, 0, 85, 136, 187, 255, 255, 255, 255, 255, 255, 255, 187, 102, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 68, 159, 221, 221, 221, 221, 221, 221, 238, 170, 119, 63, 0, 0, 102, 68, 0, 0, 51, 102, 119, 187, 170, 170, 170, 170, 170, 170, 136, 60};
    unsigned int barray[240] = {157, 168, 170, 136, 85, 0, 0, 0, 0, 0, 0, 0, 0, 85, 68, 102, 102, 0, 0, 0, 0, 0, 0, 0, 0, 119, 170, 221, 204, 187, 204, 204, 221, 187, 204, 204, 255, 255, 255, 255, 255, 255, 255, 255, 255, 204, 153, 0, 0, 0, 0, 0, 0, 17, 0, 136, 136, 170, 170, 51, 68, 51, 34, 51, 51, 51, 51, 170, 221, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 238, 204, 102, 102, 119, 119, 85, 102, 136, 136, 194, 221, 255, 221, 204, 180, 170, 153, 187, 187, 153, 153, 238, 238, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 238, 238, 153, 153, 187, 187, 153, 170, 180, 204, 221, 255, 221, 194, 136, 136, 102, 85, 119, 119, 102, 102, 204, 238, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 221, 170, 51, 51, 51, 51, 34, 51, 68, 51, 170, 170, 136, 136, 0, 17, 0, 0, 0, 0, 0, 0, 153, 204, 255, 255, 255, 255, 255, 255, 255, 255, 255, 204, 204, 187, 221, 204, 204, 187, 204, 221, 170, 119, 0, 0, 0, 0, 0, 0, 0, 0, 102, 102, 68, 85, 0, 0, 0, 0, 0, 0, 0, 0, 85, 136, 170, 168, 157};

    uint8_t y_channel[W*H];
    uint8_t u_channel[W*H];
    uint8_t v_channel[W*H];
    uint8_t u_yuv420[W*H/4];
    uint8_t v_yuv420[W*H/4];

    // (cos(alpha), cos(theta)*cos(alpha), sin(theta))
    double pi = 3.14159265;
    double alpha = 45;
    double theta = 1.5*(impl->t);
    double cos_a = cos(alpha*pi/180);
    double sin_a = sin(alpha*pi/180);
    double cos_t = cos(theta*pi/180);
    double sin_t = sin(theta*pi/180);

    unsigned int r = 0;
    unsigned int g = 0;
    unsigned int b = 0;

	// Visit every pixel of the image and assign a color generated with Perlin noise
    for(unsigned int i = 0; i < H; ++i) {     // y
        for(unsigned int j = 0; j < W; ++j) {  // x
            double x = (double)j/((double)W);
			double y = (double)i/((double)H);

            
			// Wood like structure
			double n = 20 * pn.noise(x+cos_a, y+(cos_a*2*cos_t), 2*sin_t);
			n = n - floor(n);

			// Map the values to the [0, 255] interval, for simplicity we use 
			// tones of grey
			r = floor(rarray[(impl->t)%240] * n);
			g = floor(garray[(impl->t)%240] * n);
			b = floor(barray[(impl->t)%240] * n);

            y_channel[kk] = uint8_t(floor( 0.229*double(r) + 0.587*double(g) + 0.114*double(b)));
            u_channel[kk] = uint8_t(floor(-0.169*double(r) - 0.331*double(g) + 0.500*double(b)) + 128);
            v_channel[kk] = uint8_t(floor( 0.500*double(r) - 0.419*double(g) - 0.081*double(b)) + 128);


            if(i%2==1 && j%2==1){ // y odd and x odd
                uint8_t u_mean;
                uint8_t v_mean;
                
                u_mean = uint8_t(floor((float(u_channel[(i-1)*W+j-1]) + float(u_channel[(i-1)*W+j]) + float(u_channel[i*W+j-1]) + float(u_channel[i*W+j]))/4));
                v_mean = uint8_t(floor((float(v_channel[(i-1)*W+j-1]) + float(v_channel[(i-1)*W+j]) + float(v_channel[i*W+j-1]) + float(v_channel[i*W+j]))/4));
                
                u_yuv420[nn] = u_mean;
                v_yuv420[nn] = v_mean;
                nn++;
            }
            kk++;
		}
	}
	// Save the image in a binary PPM file
    hipMemcpy(yuv, y_channel, W*H, hipMemcpyHostToDevice);
    hipMemcpy(yuv+W*H, u_yuv420, W*H/4, hipMemcpyHostToDevice);
    hipMemcpy(yuv+W*H+W*H/4, v_yuv420, W*H/4, hipMemcpyHostToDevice);
	++(impl->t);
}
