#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>


__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void init(const char *Gtext, int *Gpos, int Gtext_size, int *lastpos) 
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < Gtext_size){	
		//initialize	
		if(Gtext[idx]== '\n'){
			Gpos[idx] = 0;
			lastpos[idx] = 0;
		}
		else{
			Gpos[idx] = 1;
			lastpos[idx] = 1;
		}		
	}
}

__global__ void posParallel(int *Gpos, int Gtext_size, int *lastpos, int i, int j) 
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < Gtext_size){	
		if(idx > 0 && (idx - j >= 0))
			if(lastpos[idx] != 0 && (lastpos[idx-1] == lastpos[idx]))
				Gpos[idx] += lastpos[idx-j] ;
	}
}

__global__ void copy(int *Gpos, int Gtext_size, int *lastpos, int j) 
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < Gtext_size){	
		if(idx > 0 && (idx - j >= 0))
			lastpos[idx] = Gpos[idx];		
		
	}
}



struct nl_equal
{
    __host__ __device__
        int operator()(const char& c) {
            return c == '\n' ? 0 : 1;
        }
};

struct p_eq_l
{
    __host__ __device__
        int operator()(const int& p, const int& l){
            if(p == l && p != 0)
                return 1;
            else
                return 0;

        }
};

void CountPosition1(const char *text, int *pos, int text_size)
{
    thrust::device_ptr<const char> Gtext(text);
    thrust::device_ptr<int> Gpos(pos);
    thrust::device_ptr<int> lastpos = thrust::device_malloc<int>(text_size);
    thrust::device_ptr<int> add = thrust::device_malloc<int>(text_size);
    thrust::transform(Gtext, Gtext+text_size, Gpos, nl_equal());

    int i =0, j=0, k = 0;
	for(i=0;i<9;i++){
        thrust::copy(Gpos, Gpos+text_size-1, lastpos+1);
        lastpos[0] = 0;

        
		j = (1 << i);
        if(j < text_size){
            thrust::copy(Gpos, Gpos+text_size-j, add+j);

            for(k=0;k<j;k++)
                add[k] = 0;
        }
        
        thrust::transform(Gpos, Gpos+text_size, lastpos, lastpos, p_eq_l());
        thrust::transform(lastpos, lastpos+text_size, add, add, thrust::multiplies<int>());
        thrust::transform(Gpos, Gpos+text_size, add, Gpos, thrust::plus<int>());
	}	

}

void CountPosition2(const char *text, int *pos, int text_size)
{
	int blocksize = 1024;
    int i, j;
	int *lastpos;
	size_t poslen = text_size * sizeof(int);

	
	hipMalloc((void **) &lastpos, poslen);
    dim3 DimGrid((text_size-1)/blocksize + 1, 1, 1);
    dim3 DimBlock(blocksize, 1, 1);

	init<<<DimGrid, DimBlock>>>(text, pos, text_size, lastpos);  // initialize
	hipDeviceSynchronize();
	
	for(i=0;i<9;i++){
		j = (1 << i);
		posParallel<<<DimGrid, DimBlock>>>(pos, text_size, lastpos, i, j); 
		hipDeviceSynchronize();	
		if(i != 8){
			copy<<<DimGrid, DimBlock>>>(pos, text_size, lastpos, j); 
			hipDeviceSynchronize();		
		}
	}	
	hipFree(lastpos);

}
